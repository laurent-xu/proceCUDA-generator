#include "hip/hip_runtime.h"
#include <density/F3Grid.hh>
#include <utils/glm.hh>
#include "generation_kernel.hh"

__global__ void kernel_f3_caller(GridF3<true> grid)
{
  size_t x = blockDim.x * blockIdx.x + threadIdx.x;
  size_t y = blockDim.y * blockIdx.y + threadIdx.y;
  size_t z = blockDim.z * blockIdx.z + threadIdx.z;
  size_t dimension = grid.dim_size();

  if (x < dimension && y < dimension && z < dimension)
  {
    auto center = F3::vec3_t(16., 16., 32.);
    double radius = 10.;

    auto position = grid.to_position(x, y, z);
    F3& f3 = grid.at(x, y, z);
    f3.val = glm::distance(center, position) - radius;
    f3.grad = glm::normalize_safe(position - center);
  }
}
